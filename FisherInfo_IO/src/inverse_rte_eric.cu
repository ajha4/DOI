#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <math.h>

#include "rte.h"
#include <pthread.h>


__host__ Info_Stat * populate_info_dev();
Geometry *geom;
Phantom *phan;
Source *beam_src;
complex_double *diag_terms_host;
complex_double *sph_harm;
Info_Stat *info_stat_host;
SHORT nL;
int nTerms;

__host__ int get_vind_phanind_host(int dep, int row, int col){

  return ((geom->bounZ + dep) * (geom->nX + 2 * geom->bounX ) * (geom->nY + 2 * geom->bounY ) /* reached the correct layer */ + ( geom->bounY + row)* (geom->nX + 2 * geom->bounX ) + (geom->bounX + col));

}


__host__ Info_Stat * populate_info_dev(){

    Info_Stat *info_stat_host;
    info_stat_host = (Info_Stat *) malloc (sizeof(Info_Stat));

    info_stat_host->nX = geom->nX;
    info_stat_host->nY = geom->nY;
    info_stat_host->nZ= geom->nZ;

    info_stat_host->bounX = geom->bounX;
    info_stat_host->bounY = geom->bounY;
    info_stat_host->bounZ= geom->bounZ;

    info_stat_host->subbounX = ceilf ((geom->sub_thresh)/(geom->delX));
    info_stat_host->subbounY = ceilf ((geom->sub_thresh)/(geom->delY));
    info_stat_host->subbounZ = ceilf ((geom->sub_thresh)/(geom->delZ));

    info_stat_host->delX = geom->delX;
    info_stat_host->delY = geom->delY;
    info_stat_host->delZ= geom->delZ;

    info_stat_host->x_min = geom->x_min;
    info_stat_host->y_min = geom->y_min;
    info_stat_host->z_min = geom->z_min;

    info_stat_host->x_max = geom->x_max;
    info_stat_host->y_max = geom->y_max;
    info_stat_host->z_max = geom->z_max;

    info_stat_host->sub_thresh = geom->sub_thresh;
    info_stat_host->prop_thresh = geom->prop_thresh;
    info_stat_host->sub_vox = geom->sub_vox;
    info_stat_host->self_sub_vox = geom->self_sub_vox;

    info_stat_host->g = phan->g;
    info_stat_host->n = phan->n;
    info_stat_host->no_tiss = phan->no_tiss;

    info_stat_host->cm = C/phan->n;

    info_stat_host->no_vox = geom->no_vox;

    int i;
    for(i=0; i < phan->no_tiss; i++){
        info_stat_host->mu_tot[i] = phan->mu_abs[i] + phan->mu_sc[i];
        info_stat_host->mu_sc[i] = phan->mu_sc[i];
    }

    return info_stat_host;
}


int main(int argc, char** argv )
{
		
	time_t start_time, end_time;
	time(&start_time);
    dictionary *ini;

    complex_double *transImg;
    complex_double *gbar,*dgbar;
    complex_double *W, *W1, *out, *out2, *out3, *src, *tmp1, *df;
    

    double *g;
    double *grada,*grads;
    complex_double delg;

    char *bottomName;
    char *topName;
    char *phaseName;
    double *phaseImg;
    char numStr[100];
    char tmpStr[200];
    int n;
    FILE *gFile,*fid,*fid1,*resFID;
    int itt;
    int j,jj;
    int jnk;
    int tiss_idx;
    int numIT;
    int iip, jjp, kkp;
    int r_ind,i;
    int size;

        if (argc != 2) {
        printf("\n InverseRTE file.par\n");
        printf("     file.par is the parameter file.\n\n");
        exit(1);
    }

    // Load in the initialization file
    ini = iniparser_load(argv[1]);

    // Set up the geometry, phantom, etc
    printf("Loading in geometry and phantom information...\n");
    geom = LoadGeometry(ini);
    phan = LoadPhantom(ini,1);

    beam_src = LoadSource(ini);

	printf("Done reading source information \n");

    nL = iniparser_getint(ini,"Algorithm:nL",-1);
    nTerms = iniparser_getint(ini,"Algorithm:nTerms",1);
    bottomName = iniparser_getstring(ini, "Runtime:TransFile", NULL);
    topName = iniparser_getstring(ini, "Runtime:ReflectFile", NULL);
    phaseName = iniparser_getstring(ini,"Runtime:PhaseFile",NULL);

    numIT = iniparser_getint(ini,"Runtime:numIterations",1);

    double stepsizea, stepsizes;
    stepsizea = iniparser_getdouble(ini,"Runtime:stepsizea",1e3);
    stepsizes = iniparser_getdouble(ini,"Runtime:stepsizes",1e3);

	printf("Done reading integers \n");

	int jnk2;
    if ( (gFile = fopen(iniparser_getstring(ini,"Runtime:gFile",NULL),"r") ) == NULL){
		printf("Error in opening gfile. Exiting \n");
		exit(0);
	}
	printf("%s is gFile \n", gFile);
    fread(&jnk,sizeof(int),1,gFile);
    fread(&jnk2,sizeof(int),1,gFile);
	printf("Done reading gfile integers %d and %d \n", jnk, jnk2);
    g = (flt_doub *) malloc(sizeof(flt_doub) * geom->nX * geom->nY);
    fread(g,sizeof(flt_doub),geom->nX * geom->nY,gFile);
    fclose(gFile);

	printf("Done reading gfile \n");

#if 0
    for (i=0;i<geom->nY;i++) {
        for (j=0;j<geom->nX;j++) {
            if((g[i*geom->nX+j])!=0.0){
                   printf("%f (%d %d) \n", g[i*geom->nX+j], i,j);
	}}}
#endif

    info_stat_host = populate_info_dev();

	size = (nL+1)*(nL+1)* geom->no_vox;


    printf("Generating the spherical harmonic terms \n");
    generate_sph_harm_terms();

    unsigned int timer;
	int cnt,k;
	double tmp;
	double thresh_abs, thresh_sc;
	thresh_abs = 0.0000001;
	thresh_sc =  0.0000001;
     

	byte *flag_grada, *flag_grads;

	flag_grada = (byte*) malloc ( sizeof(byte)* phan->no_tiss);
	flag_grads = (byte*) malloc ( sizeof(byte)* phan->no_tiss);

	memset(flag_grada, 0, phan->no_tiss);
	memset(flag_grads, 0, phan->no_tiss);
 
	byte flag_net; 
	int r_ind_phan;
    resFID = fopen("Residual.out","w");
        for (itt = 0;itt < numIT;itt++ ) {

			flag_net = 1;

			for(tiss_idx = 1; tiss_idx < phan->no_tiss; tiss_idx++){
				if ( flag_grads[tiss_idx] == 0 || flag_grada[tiss_idx] == 0){
					printf("%d tissue type can still change \n", tiss_idx);
					flag_net = 0;
					break;
				}
			}
    
			if(flag_net == 1){
               printf("Time to terminate the iterations at iteration number %d \n", itt);
			   break;
			}
				
		    generate_diag_terms_host();

            sprintf(numStr,"%4d",itt);
            j = 0;
            while (numStr[j] != '\0') {
                if (numStr[j] == ' ') numStr[j] = '0';
                j++;
            }

            out2 = alloc_dist();

            src = alloc_dist();
            W = alloc_dist();

            generate_source_beam(src);

            copy_dist(src,out2);
            copy_dist(src,W);

            printf("Computing gbar\n");
            Neumann(W,out2);

            free(W);
            free(src);


            // THE GRADIENT of mua
            //___________________________________   
            gbar = generate_trans_image(out2,1);
//          OutputImage(geom,gbar,"Hi.dat");
  

#if 0 
			tmp = 0; 
			for (i=0;i<geom->nY;i++) {
		        for (j=0;j<geom->nX;j++) {
        		    if((fabs(g[i*geom->nX+j] - gbar[i*geom->nX+j].real())) > 0.000001 ){
                  		//printf("%e %e (%d %d) \n", g[i*geom->nX+j], gbar[i*geom->nX+j].real(), i,j);
					}
					tmp = tmp + g[i*geom->nX+j] - gbar[i*geom->nX+j].real();
					}}
			printf("tmp is %f \n", tmp);
#endif

            out3 = alloc_dist();

            grada = (double *)malloc(sizeof(double)*phan->no_tiss);
            printf("Compute the gradient with respect to mua");
            for (tiss_idx = 1; tiss_idx < phan->no_tiss; tiss_idx++) {

				if(flag_grada[tiss_idx] == 0){

	               copy_dist(out2,out3);

				   for (iip=0; iip<geom->nZ; iip++) {
				      for (jjp=0; jjp<geom->nY; jjp++) {
				        for (kkp=0; kkp<geom->nX; kkp++) {

	            	    r_ind = (iip + geom->bounZ)* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + (jjp + geom->bounY)* (geom->nX + 2*geom->bounX) + (kkp + geom->bounX);
						r_ind_phan =iip* geom->nX * geom->nY + jjp * geom->nX + kkp;

                    	if ( phan->tiss_type[r_ind_phan] != tiss_idx) {
						
                        	for (n=0;n<(nL+1)*(nL+1);n++) {
                            	out3[VOX_TO_SPIND(r_ind, n,(nL+1)*(nL+1))] = 0+0*I;
	                        }
    	                }
        	        }}}

            	    scale_dist(out3,-1.0*C);
                	src = alloc_dist();
	                copy_dist(out3,src);
    	            Neumann(src,out3);
        	        free(src);
            	    dgbar = generate_trans_image(out3,0);
#if 0
            		for (i=0;i<geom->nY;i++) {
		              for (j=0;j<geom->nX;j++) {
        	           if((fabs(dgbar[i*geom->nX+j].real())) > 0.00000001 && i == 4 && j == 4 && tiss_idx%(geom->nX*geom->nY) == 44){
                        //printf("g = %f, gbar =  %f, dgbar =  %f (%d %d) \n",g[j], gbar[j].real(), dgbar[i*geom->nX+j].real(), i,j);
	                    }
                    }}
#endif
                	grada[tiss_idx] = 0.0;
	                for (j = 0;j<geom->nX*geom->nY;j++) {
        	            grada[tiss_idx] = grada[tiss_idx] - 2.0*(g[j]-gbar[j].real())*dgbar[j].real();
						if ( j == 4*geom->nX + 4 && tiss_idx%(geom->nX*geom->nY) == 4*geom->nX + 4 ){
						//	printf("g = %f, gbar =  %f, dgbar =  %f \n", g[j], gbar[j].real(), dgbar[j].real()); 
						}
                	}
					//if(fabs(grada[tiss_idx]) > 0.0){
		                printf("\n mu_a tiss type:  %d of %d.  Value of grada = %e                   \n",tiss_idx,phan->no_tiss,grada[tiss_idx]);
					//}
    	            free(dgbar);
        	    }
			}
/*
            sprintf(tmpStr,"GradA%s.out",numStr);
            fid = fopen(tmpStr,"w");
            fwrite(grada,sizeof(double),phan->no_tiss,fid);
            fclose(fid);
*/
           	printf("\n");
	        free(out3);

            printf("Compute the gradient with respect to mus \n");
            grads = (double *)malloc(sizeof(double)* phan->no_tiss);
#if 1
            // THE GRADIENT of mus
            //___________________________________   


            out3 = alloc_dist();
            tmp1 = alloc_dist();

            for (tiss_idx = 1; tiss_idx < phan->no_tiss ; tiss_idx++) {


			   if(flag_grads[tiss_idx] == 0){
	                copy_dist(out2,out3);
	
    	            scale_dist(out3,-1.0*C);

        	        copy_dist(out2,tmp1);
            	    PropScatmu1(geom,phan,nL,tmp1);

    	            add_dist(tmp1,out3,out3);
				   
					for (iip=0; iip<geom->nZ; iip++) {
				      for (jjp=0; jjp<geom->nY; jjp++) {
				        for (kkp=0; kkp<geom->nX; kkp++) {

						r_ind_phan =iip* geom->nX * geom->nY + jjp * geom->nX + kkp;

                    	if ( phan->tiss_type[r_ind_phan] != tiss_idx) {
        		                for (n=0;n<(nL+1)*(nL+1);n++) {
                		            out3[VOX_TO_SPIND(r_ind, n, (nL+1)*(nL+1))] = 0+0*I;
                        		}

                    		}
	                }}}


        	        src = alloc_dist();
            	    copy_dist(out3,src);
                	Neumann(src,out3);
	                free(src);
    	            dgbar = generate_trans_image(out3,0);

        	        grads[tiss_idx] = 0.0;
            	    for (j = 0;j<geom->nX*geom->nY;j++) {
						grads[tiss_idx] = grads[tiss_idx] - 2.0*(g[j]-gbar[j].real())*dgbar[j].real();
                	}

					//if(fabs(grads[tiss_idx]) > 0.0){
		                printf("mu_s tiss_type:  %d of %d.  Value of grads= %e \n",tiss_idx,phan->no_tiss,grads[tiss_idx]);
					//}
    	            free(dgbar);
        	    }
			}

            printf("\n");
            free(out3);
            free(tmp1);
#endif

            free(out2);
		    sprintf(tmpStr,"tissmapabs%s.out",numStr);
            fid = fopen(tmpStr,"w");
		    sprintf(tmpStr,"tissmapsc%s.out",numStr);
            fid1 = fopen(tmpStr,"w");
			printf("Iteration No %d \n", itt);
            for (tiss_idx = 0;tiss_idx< phan->no_tiss;tiss_idx++) {

                if (fabs(grada[tiss_idx]) < thresh_abs)
					 flag_grada[tiss_idx] = 1;
				if ( fabs(grads[tiss_idx]) < thresh_sc)
					flag_grads[tiss_idx] = 1;

				if ( flag_grada[tiss_idx] == 0 ||  flag_grads[tiss_idx] == 0){
				printf("Old value of mua = %e mus = %e for tisstype = %d\n", phan->mu_abs[tiss_idx].real(), phan->mu_sc[tiss_idx].real(),tiss_idx);
					
				if ( flag_grada[tiss_idx] == 0){
	        	   phan->mu_abs[tiss_idx] = phan->mu_abs[tiss_idx] + stepsizea*grada[tiss_idx];
				   if(phan->mu_abs[tiss_idx].real() < 0)
						phan->mu_abs[tiss_idx] = phan->mu_abs[tiss_idx] - 2*stepsizea*grada[tiss_idx];
				}
				if ( flag_grads[tiss_idx] == 0){
    	           phan->mu_sc[tiss_idx] = phan->mu_sc[tiss_idx] + stepsizes*grads[tiss_idx];
				   if(phan->mu_sc[tiss_idx].real() < 0)
						phan->mu_sc[tiss_idx] = phan->mu_sc[tiss_idx] - 2*stepsizes*grads[tiss_idx];
				}
            
				
				printf("Value of mua grad = %e musgrad = %e for tissue type %d \n", grada[tiss_idx], grads[tiss_idx], tiss_idx);
				printf("New value of mua = %e mus = %e for tissue type = %d\n", phan->mu_abs[tiss_idx].real(), phan->mu_sc[tiss_idx].real(),tiss_idx);
				}

				fwrite(&(phan->tiss_type[j]),sizeof(byte),1,fid);
	            fwrite(&(phan->tiss_type[j]),sizeof(byte),1,fid1);

            }
    
			/*updating the absorption and scatter coeff. values in the device code */

            populate_info_dev(); 

   
            fclose(fid);
            fclose(fid1);


            free(grads);
            free(grada);

            delg = 0.0;
            for (i=0;i<geom->nX*geom->nY;i++) {
                delg = delg + (gbar[i]-g[i])*(gbar[i]-g[i]);
            }
            printf("Residual = %e\n",(delg).real());
            fwrite(&delg,sizeof(double),1,resFID);
            free(gbar);




        }

        fclose(resFID);

        free(g);
        free(beam_src);
        free(phan);
        free(geom);
        iniparser_freedict(ini);

		time(&end_time);
	
		printf("\n*------------------------------------------*\n");
		printf("\nThe total time taken by the code = %d sec \n", end_time - start_time);
		printf("\n*------------------------------------------*\n");

        return(0);
}

#if 0

void Neumann(complex_double* src_host, complex_double *out_host){
    

   unsigned int timer;
    timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    int n;
    int thread_index, tid;
    int num_layers_gpu[NUM_DEVICES];
    pthread_t thread_data[NUM_DEVICES];

    THREAD_PARAMETERS thread_parameters[NUM_DEVICES];
    int size_layer = ( geom->nX + 2*geom->bounX ) * ( geom->nY + 2*geom->bounY ) * ( nL+1) * (nL+1);

    int size = size_layer * (geom->nZ + 2*geom->bounZ);

    int num_layers_per_gpu = (int) floorf(geom->nZ / ( NUM_DEVICES));
    int rem = geom->nZ % (NUM_DEVICES);


    for (thread_index = 0; thread_index < NUM_DEVICES; thread_index++){
        num_layers_gpu[thread_index] = num_layers_per_gpu;
        if(rem > thread_index){
            num_layers_gpu[thread_index] += 1;
        }
    }
    complex_double *W_out_host;

    W_out_host = (complex_double *) malloc ( sizeof(complex_double)*size);
    memset(W_out_host, 0, sizeof(complex_double)*size);

    for (n=0;n<nTerms-1;n++) {

        for(thread_index = 0; thread_index < NUM_DEVICES; thread_index++) {
            thread_parameters[thread_index].device_index =thread_index;
            thread_parameters[thread_index].src_host = src_host;
            thread_parameters[thread_index].num_layers = num_layers_gpu[thread_index];
            thread_parameters[thread_index].layer_start = 0 ;
            for(tid = 0; tid < thread_index; tid++){
                thread_parameters[thread_index].layer_start += num_layers_gpu[tid];
            }
            thread_parameters[thread_index].out_host = W_out_host + (thread_parameters[thread_index].layer_start + geom->bounZ) * size_layer ;
            pthread_create(& thread_data[thread_index], NULL, prop_abs, &thread_parameters[thread_index]);
        }


        for(thread_index = 0; thread_index < NUM_DEVICES; thread_index++) {
            pthread_join(thread_data[thread_index], NULL);
        }

        prop_scat(W_out_host, src_host);
        add_dist( src_host, out_host, out_host);

    }

    printf("Calling the absorption kernel \n");
    cutilCheckError(cutResetTimer(timer));
    cutilCheckError(cutStartTimer(timer));

    for(thread_index = 0; thread_index < NUM_DEVICES; ++thread_index) {
        thread_parameters[thread_index].device_index = thread_index;
        thread_parameters[thread_index].src_host = out_host;
            thread_parameters[thread_index].num_layers = num_layers_gpu[thread_index];
            thread_parameters[thread_index].layer_start = 0 ;
            for(tid = 0; tid < thread_index; tid++){
                thread_parameters[thread_index].layer_start += num_layers_gpu[tid];
            }
        thread_parameters[thread_index].out_host = W_out_host + (thread_index*num_layers_per_gpu + geom->bounZ) * size_layer ;
        pthread_create(& thread_data[thread_index], NULL, prop_abs, &thread_parameters[thread_index]);
    }

    for(thread_index = 0; thread_index < NUM_DEVICES; ++thread_index) {
        pthread_join(thread_data[thread_index], NULL);
    }

   printf("Time taken for the absorption kernel :%f ms \n", cutGetTimerValue(timer));

    copy_dist(W_out_host, out_host);


	int cnt,i,j,k,r_ind;
#if 0
		   for (cnt=0;cnt< (nL+1)*(nL+1) ;cnt = cnt +2) {
			  for (i=geom->bounZ;i<geom->nZ + geom->bounZ;i++) {
			      for (j=geom->bounY;j<geom->nY + geom->bounY;j++) {
			         for (k=geom->bounX;k<geom->nX + geom->bounX;k++) {
			            r_ind = i* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + j* (geom->nX + 2*geom->bounX) + k;
            			if((out[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)]).real()!=0.0){
                        printf("% e , %e i, %d (%d %d %d) \n", out[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(), W[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(),cnt, i-geom->bounX,j-geom->bounY,k-geom->bounZ);
		    }}}}}   
#endif     
}
#endif

