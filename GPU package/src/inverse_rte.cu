#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <math.h>

#include "rte.h"
#include <pthread.h>


__host__ Info_Stat * populate_info_dev();
Geometry *geom;
Phantom *phan;
Source *beam_src;
complex_double *diag_terms_host;
complex_double *sph_harm;
Info_Stat *info_stat_host;
SHORT nL;
int nTerms;

__host__ int get_vind_phanind_host(int dep, int row, int col){

  return ((geom->bounZ + dep) * (geom->nX + 2 * geom->bounX ) * (geom->nY + 2 * geom->bounY ) /* reached the correct layer */ + ( geom->bounY + row)* (geom->nX + 2 * geom->bounX ) + (geom->bounX + col));

}


__host__ Info_Stat * populate_info_dev(){

    Info_Stat *info_stat_host;
    info_stat_host = (Info_Stat *) malloc (sizeof(Info_Stat));

    info_stat_host->nX = geom->nX;
    info_stat_host->nY = geom->nY;
    info_stat_host->nZ= geom->nZ;

    info_stat_host->bounX = geom->bounX;
    info_stat_host->bounY = geom->bounY;
    info_stat_host->bounZ= geom->bounZ;

    info_stat_host->subbounX = ceilf ((geom->sub_thresh)/(geom->delX));
    info_stat_host->subbounY = ceilf ((geom->sub_thresh)/(geom->delY));
    info_stat_host->subbounZ = ceilf ((geom->sub_thresh)/(geom->delZ));

    info_stat_host->delX = geom->delX;
    info_stat_host->delY = geom->delY;
    info_stat_host->delZ= geom->delZ;

    info_stat_host->x_min = geom->x_min;
    info_stat_host->y_min = geom->y_min;
    info_stat_host->z_min = geom->z_min;

    info_stat_host->x_max = geom->x_max;
    info_stat_host->y_max = geom->y_max;
    info_stat_host->z_max = geom->z_max;

    info_stat_host->sub_thresh = geom->sub_thresh;
    info_stat_host->prop_thresh = geom->prop_thresh;
    info_stat_host->sub_vox = geom->sub_vox;
    info_stat_host->self_sub_vox = geom->self_sub_vox;

    info_stat_host->g = phan->g;
    info_stat_host->n = phan->n;
    info_stat_host->no_tiss = phan->no_tiss;

    info_stat_host->cm = C/phan->n;

    info_stat_host->no_vox = geom->no_vox;

   /* info_stat_host->boun_blkX = ceilf(info_stat.bounX/BLK_SIZE);
    info_stat_host->boun_blkY = ceilf(info_stat.bounY/BLK_SIZE);
    info_stat_host->boun_blkZ = ceilf(info_stat.bounZ/BLK_SIZE);

    info_stat_host->boun_blk_sizeZ = info_stat_host->boun_blkZ*BLK_SIZE;
    info_stat_host->boun_blk_sizeY = info_stat_host->boun_blkY*BLK_SIZE;
    info_stat_host->boun_blk_sizeX = info_stat_host->boun_blkX*BLK_SIZE; */
   int i;
    for(i=0; i < phan->no_tiss; i++){
        info_stat_host->mu_tot[i] = phan->mu_abs[i] + phan->mu_sc[i];
        info_stat_host->mu_sc[i] = phan->mu_sc[i];
    }

    return info_stat_host;
}


int main(int argc, char** argv )
{
		
	time_t start_time, end_time;
	time(&start_time);
    dictionary *ini;

    complex_double *transImg;
    complex_double *gbar,*dgbar;
    complex_double *W, *W1, *out, *out2, *out3, *src, *tmp1, *df;
    

    double *g;
    double *grada,*grads;
    complex_double delg;

    char *bottomName;
    char *topName;
    char *phaseName;
    double *phaseImg;
    char numStr[100];
    char tmpStr[200];
    int n;
    FILE *gFile,*fid,*fid1,*resFID;
    int itt;
    int j,jj;
    int jnk;
    int idx;
    int numIT;
    int iip, jjp, kkp;
    int r_ind,i;
    int size;

        if (argc != 2) {
        printf("\n InverseRTE file.par\n");
        printf("     file.par is the parameter file.\n\n");
        exit(1);
    }

    // Load in the initialization file
    ini = iniparser_load(argv[1]);

    // Set up the geometry, phantom, etc
    printf("Loading in geometry and phantom information...\n");
    geom = LoadGeometry(ini);
    phan = LoadPhantom(ini,0);


/*
	for(j=1; j < phan->no_tiss; j++){
	    phan->mu_abs[j] = phan->mu_abs[j] - 0.005;
    	phan->mu_sc[j] = phan->mu_sc[j] - 0.1;
	}
*/
    beam_src = LoadSource(ini);

	printf("Done reading source information \n");

    nL = iniparser_getint(ini,"Algorithm:nL",-1);
    nTerms = iniparser_getint(ini,"Algorithm:nTerms",1);
    bottomName = iniparser_getstring(ini, "Runtime:TransFile", NULL);
    topName = iniparser_getstring(ini, "Runtime:ReflectFile", NULL);
    phaseName = iniparser_getstring(ini,"Runtime:PhaseFile",NULL);

    numIT = iniparser_getint(ini,"Runtime:numIterations",1);

    double stepsizea, stepsizes;
    stepsizea = iniparser_getdouble(ini,"Runtime:stepsizea",1e3);
    stepsizes = iniparser_getdouble(ini,"Runtime:stepsizes",1e3);

	printf("Done reading integers \n");

	int jnk2;
    if ( (gFile = fopen(iniparser_getstring(ini,"Runtime:gFile",NULL),"r") ) == NULL){
		printf("Error in opening gfile. Exiting \n");
		exit(0);
	}
	printf("%s is gFile \n", gFile);
    fread(&jnk,sizeof(int),1,gFile);
    fread(&jnk2,sizeof(int),1,gFile);
	printf("Done reading gfile integers %d and %d \n", jnk, jnk2);
    g = (flt_doub *) malloc(sizeof(flt_doub) * geom->nX * geom->nY);
    fread(g,sizeof(flt_doub),geom->nX * geom->nY,gFile);
    fclose(gFile);

	printf("Done reading gfile \n");

#if 0
    for (i=0;i<geom->nY;i++) {
        for (j=0;j<geom->nX;j++) {
            if((g[i*geom->nX+j])!=0.0){
                   printf("%f (%d %d) \n", g[i*geom->nX+j], i,j);
	}}}
#endif

    info_stat_host = populate_info_dev();


/*    MY_SAFE_CALL(hipMalloc(&(info_dyn_dev.mu_abs), sizeof(complex_double)*phan->no_tiss));
    MY_SAFE_CALL(hipMemcpy(info_dyn_dev.mu_abs, phan->mu_abs,sizeof(complex_double)*phan->no_tiss,hipMemcpyHostToDevice));

    MY_SAFE_CALL(hipMalloc(&(info_dyn_dev.mu_sc), sizeof(complex_double)*phan->no_tiss));
    MY_SAFE_CALL(hipMemcpy(info_dyn_dev.mu_sc, phan->mu_sc,sizeof(complex_double)*phan->no_tiss, hipMemcpyHostToDevice));
   
*/ 

	size = (nL+1)*(nL+1)* geom->no_vox;


    printf("Generating the spherical harmonic terms \n");
    generate_sph_harm_terms();

    unsigned int timer;
	int cnt,k;
	double tmp;
	double thresh_abs, thresh_sc;
	thresh_abs = 0.00001;
	thresh_sc = 0.00001; 

	byte *flag_grada, *flag_grads;

	flag_grada = (byte*) malloc ( sizeof(byte)* geom->nX*geom->nY * geom->nZ);
	flag_grads = (byte*) malloc ( sizeof(byte)* geom->nX*geom->nY * geom->nZ);

	memset(flag_grada, 0, geom->nX*geom->nY * geom->nZ);
	memset(flag_grads, 0, geom->nX*geom->nY * geom->nZ);
 
	byte flag_net; 
	int r_ind_phan;
    resFID = fopen("Residual.out","w");
        for (itt = 0;itt < numIT;itt++ ) {

			flag_net = 1;

			for(j = 0; j < geom->nX*geom->nY * geom->nZ; j++){
				if ( flag_grads[j] == 0 || flag_grada[j] == 0){
					printf("%d pixel could still be modified \n", j);
					flag_net = 0;
					break;
				}
			}
    
			if(flag_net == 1){
               printf("Time to terminate the iterations at iteration number %d \n", itt);
			   break;
			}
				
		    //printf("Generating the diagonal terms \n");
		    generate_diag_terms_host();
		    //printf("Time taken for generating the diagonal terms :%f ms \n", cutGetTimerValue(timer));

            sprintf(numStr,"%4d",itt);
            j = 0;
            while (numStr[j] != '\0') {
                if (numStr[j] == ' ') numStr[j] = '0';
                j++;
            }

            out2 = alloc_dist();

            src = alloc_dist();
            W = alloc_dist();

            generate_source_beam(src);

            copy_dist(src,out2);
            copy_dist(src,W);

            printf("Computing gbar\n");
            Neumann(W,out2);

#if 0
		   for (cnt=0;cnt< (nL+1)*(nL+1) ;cnt = cnt +2) {
			  for (i=geom->bounZ;i<geom->nZ + geom->bounZ;i++) {
			      for (j=geom->bounY;j<geom->nY + geom->bounY;j++) {
			         for (k=geom->bounX;k<geom->nX + geom->bounX;k++) {
			            r_ind = i* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + j* (geom->nX + 2*geom->bounX) + k;
            			if((out2[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)]).real()!=0.0){
                        printf("% e , %e i, %d (%d %d %d) \n", out2[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(), W[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(),cnt, i-geom->bounX,j-geom->bounY,k-geom->bounZ);
		    }}}}}   
#endif     
            free(W);
            free(src);


            // THE GRADIENT of mua
            //___________________________________   
            gbar = generate_trans_image(out2,1);
//          OutputImage(geom,gbar,"Hi.dat");
  

#if 0 
			tmp = 0; 
			for (i=0;i<geom->nY;i++) {
		        for (j=0;j<geom->nX;j++) {
        		    if((fabs(g[i*geom->nX+j] - gbar[i*geom->nX+j].real())) > 0.000001 ){
                  		//printf("%e %e (%d %d) \n", g[i*geom->nX+j], gbar[i*geom->nX+j].real(), i,j);
					}
					tmp = tmp + g[i*geom->nX+j] - gbar[i*geom->nX+j].real();
					}}
			printf("tmp is %f \n", tmp);
#endif

            out3 = alloc_dist();

            grada = (double *)malloc(sizeof(double)*geom->nX*geom->nY*geom->nZ);
            for (idx = 0; idx < geom->nX*geom->nY*geom->nZ; idx++) {


				if(flag_grada[j] == 0){

	                copy_dist(out2,out3);
	

				   for (iip=0; iip<geom->nZ; iip++) {
				      for (jjp=0; jjp<geom->nY; jjp++) {
				        for (kkp=0; kkp<geom->nX; kkp++) {

	            	    r_ind = (iip + geom->bounZ)* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + (jjp + geom->bounY)* (geom->nX + 2*geom->bounX) + (kkp + geom->bounX);
						r_ind_phan =iip* geom->nX * geom->nY + jjp * geom->nX + kkp;

                    	if ( r_ind_phan != idx) {
						
                        	for (n=0;n<(nL+1)*(nL+1);n++) {
                            	out3[VOX_TO_SPIND(r_ind, n,(nL+1)*(nL+1))] = 0+0*I;
	                        }

    	                }
//						else
//							printf("Voxel index we are working on is (%d %d %d) with idx = %d \n", iip, jjp, kkp, idx);
        	        }}}
#if 0
               for (cnt=0;cnt< (nL+1)*(nL+1) ;cnt = cnt +2) {
             	 for (i=geom->bounZ;i<geom->nZ + geom->bounZ;i++) {
                  for (j=geom->bounY;j<geom->nY + geom->bounY;j++) {
                     for (k=geom->bounX;k<geom->nX + geom->bounX;k++) {
                        r_ind = i* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + j* (geom->nX + 2*geom->bounX) + k;
                        if((out3[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)]).real()!=0.0){
                        printf("% e , %e i, %d (%d %d %d) \n", out2[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(), out3[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(),cnt, i-geom->bounX,j-geom->bounY,k-geom->bounZ);
            }}}}}   
			
#endif
			//printf("\n");

            	    scale_dist(out3,-1.0*C);
                	src = alloc_dist();
	                copy_dist(out3,src);
    	            Neumann(src,out3);
        	        free(src);
            	    dgbar = generate_trans_image(out3,0);

            		for (i=0;i<geom->nY;i++) {
		              for (j=0;j<geom->nX;j++) {
        	           if((fabs(dgbar[i*geom->nX+j].real())) > 0.00000001 && i == 4 && j == 4 && idx%(geom->nX*geom->nY) == 44){
                        //printf("g = %f, gbar =  %f, dgbar =  %f (%d %d) \n",g[j], gbar[j].real(), dgbar[i*geom->nX+j].real(), i,j);
	                    }
                    }}

                	grada[idx] = 0.0;
	                for (j = 0;j<geom->nX*geom->nY;j++) {
        	            grada[idx] = grada[idx] - 2.0*(g[j]-gbar[j].real())*dgbar[j].real();
						if ( j == 4*geom->nX + 4 && idx%(geom->nX*geom->nY) == 4*geom->nX + 4 ){
						//	printf("g = %f, gbar =  %f, dgbar =  %f \n", g[j], gbar[j].real(), dgbar[j].real()); 
						}
                	}
					if(fabs(grada[idx]) > 0.0){
		                printf("mu_a voxel:  %d of %d.  Value of grada = %e                   \r",idx,geom->nX*geom->nY*geom->nZ,grada[idx]);
					}
    	            free(dgbar);
        	    }
			}
/*
            sprintf(tmpStr,"GradA%s.out",numStr);
            fid = fopen(tmpStr,"w");
            fwrite(grada,sizeof(double),phan->no_tiss,fid);
            fclose(fid);
*/
           	printf("\n");
	        free(out3);

            grads = (double *)malloc(sizeof(double)* geom->nX*geom->nY*geom->nZ);
#if 1
            // THE GRADIENT of mus
            //___________________________________   


            out3 = alloc_dist();
            tmp1 = alloc_dist();

            for (idx = 0; idx < geom->nX * geom->nY * geom->nZ ; idx++) {


			   if(flag_grads[j] == 0){
	               copy_dist(out2,out3);
	

#if 0
               for (cnt=0;cnt< (nL+1)*(nL+1) ;cnt = cnt +2) {
             	 for (i=geom->bounZ;i<geom->nZ + geom->bounZ;i++) {
                  for (j=geom->bounY;j<geom->nY + geom->bounY;j++) {
                     for (k=geom->bounX;k<geom->nX + geom->bounX;k++) {
                        r_ind = i* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + j* (geom->nX + 2*geom->bounX) + k;
                        if((out2[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)]).real()!=0.0){
                        printf("% e , %e i, %d (%d %d %d) \n", out2[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(), out3[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(),cnt, i-geom->bounX,j-geom->bounY,k-geom->bounZ);
            }}}}}   
#endif

    	            scale_dist(out3,-1.0*C);

        	        copy_dist(out2,tmp1);
            	    PropScatmu1(geom,phan,nL,tmp1);

    	            add_dist(tmp1,out3,out3);
				   
					for (iip=0; iip<geom->nZ; iip++) {
				      for (jjp=0; jjp<geom->nY; jjp++) {
				        for (kkp=0; kkp<geom->nX; kkp++) {

	            	    r_ind = (iip + geom->bounZ)* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + (jjp + geom->bounY)* (geom->nX + 2*geom->bounX) + (kkp + geom->bounX);
						r_ind_phan =iip* geom->nX * geom->nY + jjp * geom->nX + kkp;

                    	if ( r_ind_phan != idx) {
        		                for (n=0;n<(nL+1)*(nL+1);n++) {
                		            out3[VOX_TO_SPIND(r_ind, n, (nL+1)*(nL+1))] = 0+0*I;
                        		}

                    		}
	                }}}


        	        src = alloc_dist();
            	    copy_dist(out3,src);
                	Neumann(src,out3);
	                free(src);
    	            dgbar = generate_trans_image(out3,0);

        	        grads[idx] = 0.0;
            	    for (j = 0;j<geom->nX*geom->nY;j++) {
						grads[idx] = grads[idx] - 2.0*(g[j]-gbar[j].real())*dgbar[j].real();
                	}

					if(fabs(grads[idx]) > 0.0){
		                printf("mu_s voxel:  %d of %d.  Value of grads= %e \r",idx,geom->nX*geom->nY*geom->nZ,grads[idx]);
					}
    	            free(dgbar);
        	    }
			}
/*
            sprintf(tmpStr,"GradS%s.out",numStr);
            fid = fopen(tmpStr,"w");
            fwrite(grads,sizeof(double),phan->no_tiss,fid);
            fclose(fid);
*/
            printf("\n");
            free(out3);
            free(tmp1);
#endif



            //
            free(out2);
		    sprintf(tmpStr,"tissmapabs%s.out",numStr);
            fid = fopen(tmpStr,"w");
		    sprintf(tmpStr,"tissmapsc%s.out",numStr);
            fid1 = fopen(tmpStr,"w");
			printf("Iteration No %d \n", itt);
            for (j = 0;j<geom->nX * geom->nY * geom->nZ;j++) {

                if (fabs(grada[j]) < thresh_abs)
					 flag_grada[j] = 1;
				if ( fabs(grads[j]) < thresh_sc)
					flag_grads[j] = 1;


				r_ind = get_vind_phanind_host(j/(geom->nX*geom->nY), (j/(geom->nX)) % (geom->nY) ,j%(geom->nX*geom->nY));
				if ( flag_grada[j] == 0 ||  flag_grads[j] == 0){
				printf("Old value of mua = %e mus = %e for pixel %d tisstype = %d\n", phan->mu_abs[phan->tiss_type[r_ind]].real(), phan->mu_sc[phan->tiss_type[j]].real(),j,phan->tiss_type[j]);
					
				if ( flag_grada[j] == 0){
					if ( phan->tiss_type[r_ind] - 2*SIGN(grada[j]) < phan->no_tiss && (phan->tiss_type[r_ind] - 2*SIGN(grada[j])) > 0 && itt%2 == 0)  {
	        	        phan->tiss_type[r_ind] = phan->tiss_type[r_ind] - 2*SIGN(grada[j]);
					}
				}
				if ( flag_grads[j] == 0){
					if ( phan->tiss_type[r_ind] - 2*SIGN(grads[j]) < phan->no_tiss && (phan->tiss_type[r_ind] - 2*SIGN(grads[j])) > 0 && itt%2 == 1)  {
    	            	phan->tiss_type[r_ind] = phan->tiss_type[r_ind] - 2*SIGN(grads[j]);
					}
				}
            
				
				printf("Value of mua grad = %e musgrad = %e for pixel %d \n", grada[j], grads[j], j);
				printf("New value of mua = %e mus = %e for pixel %d tisstype = %d\n", phan->mu_abs[phan->tiss_type[r_ind]].real(), phan->mu_sc[phan->tiss_type[r_ind]].real(), j,phan->tiss_type[r_ind]);
				}

				fwrite(&(phan->tiss_type[j]),sizeof(byte),1,fid);
	            fwrite(&(phan->tiss_type[j]),sizeof(byte),1,fid1);

            }
    
			/*updating the absorption and scatter coeff. values in the device code */ 

   
            fclose(fid);
            fclose(fid1);


            free(grads);
            free(grada);

            delg = 0.0;
            for (i=0;i<geom->nX*geom->nY;i++) {
                delg = delg + (gbar[i]-g[i])*(gbar[i]-g[i]);
            }
            printf("Residual = %e\n",(delg).real());
            fwrite(&delg,sizeof(double),1,resFID);
            free(gbar);




        }

        fclose(resFID);

        free(g);
        free(beam_src);
        free(phan);
        free(geom);
        iniparser_freedict(ini);

		time(&end_time);
	
		printf("\n*------------------------------------------*\n");
		printf("\nThe total time taken by the code = %d sec \n", end_time - start_time);
		printf("\n*------------------------------------------*\n");

        return(0);
}

#if 0

void Neumann(complex_double* src_host, complex_double *out_host){
    

   unsigned int timer;
    timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    int n;
    int thread_index, tid;
    int num_layers_gpu[NUM_DEVICES];
    pthread_t thread_data[NUM_DEVICES];

    THREAD_PARAMETERS thread_parameters[NUM_DEVICES];
    int size_layer = ( geom->nX + 2*geom->bounX ) * ( geom->nY + 2*geom->bounY ) * ( nL+1) * (nL+1);

    int size = size_layer * (geom->nZ + 2*geom->bounZ);

    int num_layers_per_gpu = (int) floorf(geom->nZ / ( NUM_DEVICES));
    int rem = geom->nZ % (NUM_DEVICES);


    for (thread_index = 0; thread_index < NUM_DEVICES; thread_index++){
        num_layers_gpu[thread_index] = num_layers_per_gpu;
        if(rem > thread_index){
            num_layers_gpu[thread_index] += 1;
        }
    }
    complex_double *W_out_host;

    W_out_host = (complex_double *) malloc ( sizeof(complex_double)*size);
    memset(W_out_host, 0, sizeof(complex_double)*size);

    for (n=0;n<nTerms-1;n++) {

        for(thread_index = 0; thread_index < NUM_DEVICES; thread_index++) {
            thread_parameters[thread_index].device_index =thread_index;
            thread_parameters[thread_index].src_host = src_host;
            thread_parameters[thread_index].num_layers = num_layers_gpu[thread_index];
            thread_parameters[thread_index].layer_start = 0 ;
            for(tid = 0; tid < thread_index; tid++){
                thread_parameters[thread_index].layer_start += num_layers_gpu[tid];
            }
            thread_parameters[thread_index].out_host = W_out_host + (thread_parameters[thread_index].layer_start + geom->bounZ) * size_layer ;
            pthread_create(& thread_data[thread_index], NULL, prop_abs, &thread_parameters[thread_index]);
        }


        for(thread_index = 0; thread_index < NUM_DEVICES; thread_index++) {
            pthread_join(thread_data[thread_index], NULL);
        }

        prop_scat(W_out_host, src_host);
        add_dist( src_host, out_host, out_host);

    }

    printf("Calling the absorption kernel \n");
    cutilCheckError(cutResetTimer(timer));
    cutilCheckError(cutStartTimer(timer));

    for(thread_index = 0; thread_index < NUM_DEVICES; ++thread_index) {
        thread_parameters[thread_index].device_index = thread_index;
        thread_parameters[thread_index].src_host = out_host;
            thread_parameters[thread_index].num_layers = num_layers_gpu[thread_index];
            thread_parameters[thread_index].layer_start = 0 ;
            for(tid = 0; tid < thread_index; tid++){
                thread_parameters[thread_index].layer_start += num_layers_gpu[tid];
            }
        thread_parameters[thread_index].out_host = W_out_host + (thread_index*num_layers_per_gpu + geom->bounZ) * size_layer ;
        pthread_create(& thread_data[thread_index], NULL, prop_abs, &thread_parameters[thread_index]);
    }

    for(thread_index = 0; thread_index < NUM_DEVICES; ++thread_index) {
        pthread_join(thread_data[thread_index], NULL);
    }

   printf("Time taken for the absorption kernel :%f ms \n", cutGetTimerValue(timer));

    copy_dist(W_out_host, out_host);


	int cnt,i,j,k,r_ind;
#if 0
		   for (cnt=0;cnt< (nL+1)*(nL+1) ;cnt = cnt +2) {
			  for (i=geom->bounZ;i<geom->nZ + geom->bounZ;i++) {
			      for (j=geom->bounY;j<geom->nY + geom->bounY;j++) {
			         for (k=geom->bounX;k<geom->nX + geom->bounX;k++) {
			            r_ind = i* (geom->nX + 2*geom->bounX )* (geom->nY + 2*geom->bounY) + j* (geom->nX + 2*geom->bounX) + k;
            			if((out[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)]).real()!=0.0){
                        printf("% e , %e i, %d (%d %d %d) \n", out[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(), W[VOX_TO_SPIND(r_ind,cnt,geom->no_vox)].real(),cnt, i-geom->bounX,j-geom->bounY,k-geom->bounZ);
		    }}}}}   
#endif     
}
#endif

